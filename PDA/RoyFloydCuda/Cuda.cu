#include "hip/hip_runtime.h"
#include<iostream>
#include "hip/hip_runtime.h"
#include ""
using namespace std;

#define N 5
#define INF 99999


// Device code
__global__ void RoyFloyd(int Matrice[N][N])
{
	int i = threadIdx.x;
	int j = threadIdx.y;

    if(A[procid][k] + A[k][j] < A[procid][j])
		A[procid][j] = A[procid][k] + A[k][j];
}
            
// Host code
int main()
{
   size_t size = N * N * sizeof(int);

    // Allocate input vectors h_A and h_B in host memory

    // Initialize input vectors
	int h_A[N][N] = {
		0,3,9,8,3,
		5,0,1,4,2,
		6,6,0,4,5,
		2,9,2,0,7,
		7,9,3,2,0,
	};
	
    // Allocate vectors in device memory
    int* d_A;
    hipMalloc(&d_A, size);
	
    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    // Invoke kernel
    dim3 threadsPerBlock(N, N);
	int numBlocks= 1;

	for(int k = 0; k < N; k++)
	{
		VecAdd<<<numBlocks, threadsPerBlock>>>(d_A, k);

	}
		
    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
            
    // Free host memory
    for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			if (h_A[i][j] == INF)
				cout << "INF ";
			else
				cout << h_A[i][j] << " ";
		}
		cout << endl;
}

}